#include "hip/hip_runtime.h"
#include "constants.h"
#include "cuda_kernels.cuh"
#include "models/models.cuh"
#include "estimators/estimators.cuh"

/* Description of the cuda_calc_curve_values function
* ===================================================
*
* This function calls one of the fitting curve functions depending on the input
* parameter model_id. The fitting curve function calculates the values of
* the fitting curves and its partial derivatives with respect to the fitting
* curve parameters. Multiple fits are calculated in parallel.
*
* Parameters:
*
* parameters: An input vector of concatenated sets of model parameters.
*
* n_fits: The number of fits.
*
* n_points: The number of data points per fit.
*
* n_parameters: The number of curve parameters.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* values: An output vector of concatenated sets of model function values.
*
* derivatives: An output vector of concatenated sets of model function partial
*              derivatives.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* n_blocks_per_fit: The number of thread blocks used to calculate one fit.
*
* model_id: The fitting model ID.
*
* chunk_index: The data chunk index.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calc_curve_values function
* ===========================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   cuda_calc_curve_values<<< blocks, threads >>>(
*       parameters,
*       n_fits,
*       n_points,
*       n_parameters,
*       finished,
*       values,
*       derivatives,
*       n_fits_per_block,
*       n_blocks_per_fit,
*       model_id,
*       chunk_index,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calc_curve_values(
    REAL const * parameters,
    int const n_fits,
    int const n_points,
    int const n_parameters,
    int const * finished,
    REAL * values,
    REAL * derivatives,
    int const n_fits_per_block,
    int const n_blocks_per_fit,
    ModelID const model_id,
    int const chunk_index,
    char * user_info,
    std::size_t const user_info_size)
{
    int const fit_in_block = threadIdx.x / n_points;
    int const fit_index = blockIdx.x * n_fits_per_block / n_blocks_per_fit + fit_in_block;
    int const fit_piece = blockIdx.x % n_blocks_per_fit;
    int const point_index = threadIdx.x - fit_in_block * n_points + fit_piece * blockDim.x;
    int const first_point = fit_index * n_points;

    REAL * current_values = values + first_point;
    REAL * current_derivatives = derivatives + first_point * n_parameters;
    REAL const * current_parameters = parameters + fit_index * n_parameters;

    if (finished[fit_index])
        return;
    if (point_index >= n_points)
        return;

    calculate_model(model_id, current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size);
}

/* Description of the sum_up_floats function
* ==========================================
*
* This function sums up a vector of REAL values and stores the result at the
* first place of the vector.
*
* Parameters:
*
* shared_array: An input vector of REAL values. The vector must be stored
*               on the shared memory of the GPU. The size of this vector must be a
*               power of two. Use zero padding to extend it to the next highest
*               power of 2 greater than the number of elements.
*
* size: The number of elements in the input vector considering zero padding.
*
* Calling the sum_up_floats function
* ==================================
*
* This __device__ function can be only called from a __global__ function or
* an other __device__ function. When calling the function, the blocks and threads
* of the __global__ function must be set up correctly, as shown in the following
* example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = size * vectors_per_block;
*   blocks.x = n_vectors / vectors_per_block;
*
*   global_function<<< blocks, threads >>>(parameter1, ...);
*
*/

__device__ void sum_up_floats(volatile REAL* shared_array, int const size)
{
    int const fit_in_block = threadIdx.x / size;
    int const point_index = threadIdx.x - (fit_in_block*size);

    int current_n_points = size >> 1;
    __syncthreads();
    while (current_n_points)
    {
        if (point_index < current_n_points)
        {
            shared_array[point_index] += shared_array[point_index + current_n_points];
        }
        current_n_points >>= 1;
        __syncthreads();
    }
}

/* Description of the cuda_sum_chi_square_subtotals function
* ==========================================================
*
* This function sums up chi_square subtotals in place.
*
* Parameters:
*
* chi_squares: A vector of chi-square values for multiple fits.
*              in: subtotals
*              out: totals
*
* n_blocks_per_fit: The number of blocks used to calculate one fit. It is
*                   equivalent to the number of subtotals per fit.
*
* n_fits: The number of fits.
*
* finished: An input vector which allows the calculation to be skipped
*           for single fits.
*
* Calling the cuda_sum_chi_square_subtotals function
* ==================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_sum_chi_square_subtotals<<< blocks, threads >>>(
*       chi_squares,
*       n_blocks_per_fit,
*       n_fits,
*       finished);
*
*/

__global__ void cuda_sum_chi_square_subtotals(
    REAL * chi_squares,
    REAL const * subtotals,
    int const n_blocks_per_fit,
    int const n_fits,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n_fits || finished[index])
        return;

    REAL * chi_square = chi_squares + index;
    REAL const * subtotal = subtotals + index;

    double sum = 0.0;
    for (int i = 0; i < n_blocks_per_fit; i++)
        sum += subtotal[i * n_fits];

    chi_square[0] = sum;
}

/* Description of the cuda_check_fit_improvement function
* =======================================================
*
* This function checks after each calculation of chi-square values whether the
* currently calculated chi-square values are lower than chi-square values calculated
* in the previous iteration and sets the iteration_failed flags.
*
* Parameters:
*
* iteration_failed: An output vector of flags which indicate whether the fitting
*                   process improved the fit in the last iteration. If yes it is set
*                   to 0 otherwise to 1.
*
* chi_squares: An input vector of chi-square values for multiple fits.
*
* prev_chi_squares: An input vector of chi-square values for multiple fits calculated
*                   in the previous iteration.
*
* n_fits: The number of fits.
*
* finished: An input vector which allows the calculation to be skipped
*           for single fits.
*
* Calling the cuda_check_fit_improvement function
* ===============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_check_fit_improvement <<< blocks, threads >>>(
*       iteration_failed,
*       chi_squares,
*       prev_chi_squares,
*       n_fits,
*       finished);
*
*/

__global__ void cuda_check_fit_improvement(
    int * iteration_failed,
    REAL const * chi_squares,
    REAL const * prev_chi_squares,
    int const n_fits,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n_fits || finished[index])
        return;

    bool const prev_chi_squares_initialized = prev_chi_squares[index] != 0.;
    // chi_squares[index] can be NaN which compares to false with any other number
    bool const chi_square_decreased = (chi_squares[index] < prev_chi_squares[index]);
    if (prev_chi_squares_initialized && !chi_square_decreased)
    {
        iteration_failed[index] = 1;
    }
    else
    {
        iteration_failed[index] = 0;
    }
}

/* Description of the cuda_calculate_chi_squares function
* ========================================================
*
* This function calls one of the estimator funktions depending on the input
* parameter estimator_id. The estimator function calculates the chi-square values.
* The calcluation is performed for multiple fits in parallel.
*
* Parameters:
*
* chi_squares: An output vector of concatenated chi-square values.
*
* states: An output vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred. In this function
*         it is only used for MLE. It is set to 3 if a fitting curve value is
*         negative. This vector includes the states for multiple fits.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_points: The number of data points per fit.
*
* n_fits: The number of fits.
*
* estimator_id: The estimator ID.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calculate_chi_squares function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = power_of_two_n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   int const shared_size = sizeof(REAL) * threads.x;
*
*   cuda_calculate_chi_squares<<< blocks, threads, shared_size >>>(
*       chi_squares,
*       states,
*       data,
*       values,
*       weights,
*       n_points,
*       n_fits,
*       estimator_id,
*       finished,
*       n_fits_per_block,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_chi_squares(
    REAL * chi_squares,
    int * states,
    REAL const * data,
    REAL const * values,
    REAL const * weights,
    int const n_points,
    int const n_fits,
    int const estimator_id,
    int const * finished,
    int const n_fits_per_block,
    char * user_info,
    std::size_t const user_info_size)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const fit_piece = blockIdx.x / n_fits;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block - fit_piece * n_fits;
    int const point_index = threadIdx.x - fit_in_block * shared_size + fit_piece * shared_size;
    int const first_point = fit_index * n_points;

    if (finished[fit_index])
    {
        return;
    }

    REAL const * current_data = &data[first_point];
    REAL const * current_weight = weights ? &weights[first_point] : NULL;
    REAL const * current_value = &values[first_point];
    int * current_state = &states[fit_index];

    extern __shared__ REAL extern_array[];

    volatile REAL * shared_chi_square
        = extern_array + (fit_in_block - fit_piece) * shared_size;

    if (point_index >= n_points)
    {
        shared_chi_square[point_index] = 0.;
    }

    if (point_index < n_points)
    {
        calculate_chi_square(
            estimator_id,
            shared_chi_square,
            point_index,
            current_data,
            current_value,
            current_weight,
            current_state,
            user_info,
            user_info_size);
    }
    shared_chi_square += fit_piece * shared_size;
    sum_up_floats(shared_chi_square, shared_size);
    chi_squares[fit_index + fit_piece * n_fits] = shared_chi_square[0];
}

/* Description of the cuda_sum_gradient_subtotals function
* ========================================================
*
* This function sums up the chi-square gradient subtotals in place.
*
* Parameters:
*
* gradients: A vector of gradient values for multiple fits.
*            in: subtotals
*            out: totals
*
* n_blocks_per_fit: The number of blocks used to calculate one fit
*
* n_fits: The number of fits.
*
* n_parameters_to_fit: The number of model parameters, that are not held fixed.
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* Calling the cuda_sum_gradient_subtotals function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_sum_gradient_subtotals<<< blocks,threads >>>(
*       gradients,
*       n_blocks_per_fit,
*       n_fits,
*       n_parameters_to_fit,
*       skip,
*       finished);
*
*/

__global__ void cuda_sum_gradient_subtotals(
    REAL * gradients,
    REAL const * subtotals,
    int const n_blocks_per_fit,
    int const n_fits,
    int const n_parameters,
    int const * skip,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;
    int const fit_index = index / n_parameters;

    if (fit_index >= n_fits || finished[fit_index] || skip[fit_index])
        return;

    REAL * gradient = gradients + index;
    REAL const * subtotal = subtotals + index;

    double sum = 0.0;
    for (int i = 0; i < n_blocks_per_fit; i++)
        sum += subtotal[i * n_fits * n_parameters];

    gradient[0] = sum;
}

/* Description of the cuda_calculate_gradients function
* =====================================================
*
* This function calls one of the gradient functions depending on the input
* parameter estimator_id. The gradient function calculates the gradient values
* of the chi-square function calling a __device__ function. The calcluation is
* performed for multiple fits in parallel.
*
* Parameters:
*
* gradients: An output vector of concatenated sets of gradient vector values.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* derivatives: An input vector of concatenated sets of model function partial
*              derivatives.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_points: The number of data points per fit.
*
* n_fits: The number of fits.
*
* n_parameters: The number of fitting curve parameters.
*
* n_parameters_to_fit: The number of fitting curve parameters, that are not held
*                      fixed.
*
* parameters_to_fit_indices: An input vector of indices of fitting curve parameters,
*                            that are not held fixed.
*
* estimator_id: The estimator ID.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* user_info: An input vector containing user information.
*
* user_info_size: The number of elements in user_info.
*
* Calling the cuda_calculate_gradients function
* =============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = power_of_two_n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   int const shared_size = sizeof(REAL) * threads.x;
*
*   cuda_calculate_gradients<<< blocks, threads, shared_size >>>(
*       gradients,
*       data,
*       values,
*       derivatives,
*       weight,
*       n_points,
*       n_fits,
*       n_parameters,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       estimator_id,
*       finished,
*       skip,
*       n_fits_per_block,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_gradients(
    REAL * gradients,
    REAL const * data,
    REAL const * values,
    REAL const * derivatives,
    REAL const * weights,
    int const n_points,
    int const n_fits,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const estimator_id,
    int const * finished,
    int const * skip,
    int const n_fits_per_block,
    char * user_info,
    std::size_t const user_info_size)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const fit_piece = blockIdx.x / n_fits;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block - fit_piece * n_fits;
    int const point_index = threadIdx.x - fit_in_block * shared_size + fit_piece * shared_size;
    int const first_point = fit_index * n_points;

    if (finished[fit_index] || skip[fit_index])
    {
        return;
    }

    REAL const * current_data = &data[first_point];
    REAL const * current_weight = weights ? &weights[first_point] : NULL;
    REAL const * current_derivative = &derivatives[first_point * n_parameters];
    REAL const * current_value = &values[first_point];

    extern __shared__ REAL extern_array[];

    volatile REAL * shared_gradient = extern_array + (fit_in_block - fit_piece) * shared_size;

    if (point_index >= n_points)
    {
        shared_gradient[point_index] = 0.;
    }

    for (int parameter_index = 0; parameter_index < n_parameters_to_fit; parameter_index++)
    {
        if (point_index < n_points)
        {
            int const derivative_index = parameters_to_fit_indices[parameter_index] * n_points + point_index;

            calculate_gradient(
                estimator_id,
                shared_gradient,
                point_index,
                derivative_index,
                current_data,
                current_value,
                current_derivative,
                current_weight,
                user_info,
                user_info_size);
        }
        sum_up_floats(shared_gradient + fit_piece * shared_size, shared_size);
        gradients[(fit_index * n_parameters_to_fit + parameter_index) + fit_piece * n_fits * n_parameters_to_fit]
            = shared_gradient[fit_piece * shared_size];
    }
}

/* Description of the cuda_calculate_hessians function
* ====================================================
*
* This function calls one of the hessian function depending on the input
* parameter estimator_id. The hessian funcion calculates the hessian matrix
* values of the chi-square function calling a __device__ functions. The
* calcluation is performed for multiple fits in parallel.
*
* Parameters:
*
* hessians: An output vector of concatenated sets of hessian matrix values.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* derivatives: An input vector of concatenated sets of model function partial
*              derivatives.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_fits: The number of fits.
*
* n_points: The number of data points per fit.
*
* n_parameters: The number of fitting curve parameters.
*
* n_parameters_to_fit: The number of fitting curve parameters, that are not held
*                      fixed.
*
* parameters_to_fit_indices: An input vector of indices of fitting curve parameters,
*                            that are not held fixed.
*
* estimator_id: The estimator ID.
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calculate_hessians function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int n_unique_values = n_parameters_to_fit * (n_parameters_to_fit + 1) / 2;
*
*   threads.x
*       = min(n_unique_values * n_fits_per_block, max_threads_per_block);
*
*   blocks.y
*       = threads.x / max_threads_per_block
*       + int((threads.x % max_threads_per_block) > 0);
*
*   blocks.x
*       = n_fits / n_fits_per_block
*       + int((n_fits % n_fits_per_block) > 0);
*
*   cuda_calculate_hessians<<< blocks, threads >>>(
*       hessians,
*       data,
*       values,
*       derivatives,
*       weight,
*       n_fits,
*       n_points,
*       n_parameters,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       estimator_id,
*       skip,
*       finished,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_hessians(
    REAL * hessians,
    REAL const * data,
    REAL const * values,
    REAL const * derivatives,
    REAL const * weights,
    int const n_fits,
    int const n_points,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const estimator_id,
    int const * skip,
    int const * finished,
    char * user_info,
    std::size_t const user_info_size)
{
    int const n_unique_values = n_parameters_to_fit * (n_parameters_to_fit + 1) / 2;
    int const n_fits_per_block = blockDim.x * gridDim.y / n_unique_values;
    
    int const fit_in_block
        = (gridDim.y == 1)
        ? (blockIdx.y * blockDim.x + threadIdx.x) / n_unique_values
        : 0;

    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;

    if (fit_index >= n_fits || finished[fit_index] || skip[fit_index])
    {
        return;
    }

    int const first_point = fit_index * n_points;
    int const parameter_index = (blockIdx.y * blockDim.x + threadIdx.x) - fit_in_block * n_unique_values;

    if (parameter_index >= n_unique_values)
    {
        return;
    }

    int const parameter_index_i
        = n_parameters_to_fit
        - 1.
        - std::floor(
            .5*(
                std::sqrt(
                    - 8. * (parameter_index - n_parameters_to_fit)
                    + 4. * n_parameters_to_fit * (n_parameters_to_fit - 1.)
                    - 7.
                ) - 1.
            )
        );

    int const parameter_index_j
        = parameter_index
        + parameter_index_i
        - parameter_index_i*(n_parameters_to_fit - (parameter_index_i - 1) / 2.);

    REAL * current_hessian = &hessians[fit_index * n_parameters_to_fit * n_parameters_to_fit];
    REAL const * current_data = &data[first_point];
    REAL const * current_weight = weights ? &weights[first_point] : NULL;
    REAL const * current_derivative = &derivatives[first_point*n_parameters];
    REAL const * current_value = &values[first_point];

    int const hessian_index_ij = parameter_index_i * n_parameters_to_fit + parameter_index_j;
    int const hessian_index_ji = parameter_index_j * n_parameters_to_fit + parameter_index_i;
    int const derivative_index_i = parameters_to_fit_indices[parameter_index_i] * n_points;
    int const derivative_index_j = parameters_to_fit_indices[parameter_index_j] * n_points;

    double sum = 0.0;
    for (int point_index = 0; point_index < n_points; point_index++)
    {
        calculate_hessian(
            estimator_id,
            &sum,
            point_index,
            derivative_index_i + point_index,
            derivative_index_j + point_index,
            current_data,
            current_value,
            current_derivative,
            current_weight,
            user_info,
            user_info_size);
    }
    current_hessian[hessian_index_ij] = sum;
    current_hessian[hessian_index_ji] = sum;
}

/* Description of the cuda_modify_step_widths function
* ====================================================
*
* This function midifies the diagonal elements of the hessian matrices by multiplying
* them by the factor (1+ lambda). This operation controls the step widths of the
* iteration. If the last iteration failed, befor modifying the hessian, the diagonal
* elements of the hessian are calculated back to represent unmodified values.
*
* hessians: An input and output vector of hessian matrices, which are modified by
*           the lambda values.
*
* lambdas: An input vector of values for modifying the hessians.
*
* n_parameters: The number of fitting curve parameters.
*
* iteration_failed: An input vector which indicates whether the previous iteration
*                   failed.
*
* finished: An input vector which allows the calculation to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* Calling the cuda_modify_step_widths function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_parameters_to_fit * n_fits_per_block;
*   blocks.x = n_fits / n_fits_per_block;
*
*   cuda_modify_step_width<<< blocks, threads >>>(
*       hessians,
*       lambdas,
*       n_parameters,
*       iteration_failed,
*       finished,
*       n_fits_per_block);
*
*/

__global__ void cuda_modify_step_widths(
    REAL * hessians,
    REAL const * lambdas,
    REAL * scaling_vectors,
    unsigned int const n_parameters,
    int const * iteration_failed,
    int const * finished,
    int const n_fits_per_block)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const parameter_index = threadIdx.x - fit_in_block * shared_size;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;

    if (finished[fit_index])
    {
        return;
    }

    REAL * hessian = &hessians[fit_index * n_parameters * n_parameters];
    REAL * scaling_vector = &scaling_vectors[fit_index * n_parameters];
    REAL const & lambda = lambdas[fit_index];

    int const diagonal_index = parameter_index * n_parameters + parameter_index;

    if (iteration_failed[fit_index])
    {
        hessian[diagonal_index] -= scaling_vector[parameter_index] * lambda / 10.;
    }

    // adaptive scaling
    scaling_vector[parameter_index]
        = max(scaling_vector[parameter_index], hessian[diagonal_index]);

    // continuous scaling
    //scaling_vector[parameter_index] = hessian[diagonal_index];
    
    // initial scaling
    //if (scaling_vector[parameter_index] == 0.)
    //    scaling_vector[parameter_index] = hessian[diagonal_index];

    hessian[diagonal_index] += scaling_vector[parameter_index] * lambda;
}

__device__ void project_parameter_to_box(REAL & parameter, REAL const lower_bound, REAL const upper_bound, int const constraint_type)
{
    switch (constraint_type)
    {
    case ConstraintType::LOWER:
        parameter = max(parameter, lower_bound);
        break;
    case ConstraintType::UPPER:
        parameter = min(parameter, upper_bound);
        break;
    case ConstraintType::LOWER_UPPER:
        parameter = max(parameter, lower_bound);
        parameter = min(parameter, upper_bound);
        break;
    default:
        break;
    }
}

__global__ void cuda_project_parameters_to_box(
    REAL * parameters,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    REAL const * constraints,
    int const * constraint_types,
    int const * finished,
    int const n_fits_per_block)
{
    int const fit_in_block = threadIdx.x / n_parameters_to_fit;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;
    int const parameter_index = threadIdx.x - fit_in_block * n_parameters_to_fit;

    if (finished[fit_index])
    {
        return;
    }

    REAL & parameter = parameters[fit_index * n_parameters + parameters_to_fit_indices[parameter_index]];

    REAL const  & lower_bound = constraints[parameters_to_fit_indices[parameter_index] * 2 + LOWER_BOUND];
    REAL const  & upper_bound = constraints[parameters_to_fit_indices[parameter_index] * 2 + UPPER_BOUND];
    
    int const & constraint_type = constraint_types[parameters_to_fit_indices[parameter_index]];

    project_parameter_to_box(parameter, lower_bound, upper_bound, constraint_type);
}

/* Description of the cuda_update_parameters function
* ===================================================
*
* This function stores the fitting curve parameter values in prev_parameters and
* updates them after each iteration.
*
* Parameters:
*
* parameters: An input and output vector of concatenated sets of model
*             parameters.
*
* prev_parameters: An input and output vector of concatenated sets of model
*                  parameters calculated by the previous iteration.
*
* deltas: An input vector of concatenated delta values, which are added to the
*         model parameters.
*
* n_parameters_to_fit: The number of fitted curve parameters.
*
* parameters_to_fit_indices: The indices of fitted curve parameters.
*
* finished: An input vector which allows the parameter update to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each threadblock.
*
* Calling the cuda_update_parameters function
* ===========================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_parameters * n_fits_per_block;
*   blocks.x = n_fits / n_fits_per_block;
*
*   cuda_update_parameters<<< blocks, threads >>>(
*       parameters,
*       prev_parameters,
*       deltas,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       finished,
*       n_fits_per_block);
*
*/

__global__ void cuda_update_parameters(
    REAL * parameters,
    REAL * prev_parameters,
    REAL const * deltas,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const * finished,
    int const n_fits_per_block)
{
    int const n_parameters = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / n_parameters;
    int const parameter_index = threadIdx.x - fit_in_block * n_parameters;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;

    REAL * current_parameters = &parameters[fit_index * n_parameters];
    REAL * current_prev_parameters = &prev_parameters[fit_index * n_parameters];

    current_prev_parameters[parameter_index] = current_parameters[parameter_index];

    if (finished[fit_index])
    {
        return;
    }

    if (parameter_index >= n_parameters_to_fit)
    {
        return;
    }

    REAL const * current_deltas = &deltas[fit_index * n_parameters_to_fit];

    current_parameters[parameters_to_fit_indices[parameter_index]] += current_deltas[parameter_index];
}

/* Description of the cuda_update_state_after_solving function
 * ===========================================================
 *
 * This function interprets the singular flag vector of the equation system
 * solving function according to this LM implementation.
 *
 * Parameters:
 *
 * n_fits: The number of fits.
 *
 * solution_info: An input vector used to report whether a fit is singular.
 *
 * finished: An input vector which allows the calculation to by skipped for
 *           single fits.
 *
 * gpufit_states: An output vector of values which indicate whether the fitting
 *                process was carreid out correctly or which problem occurred.
 *                If a hessian matrix of a fit is singular, it is set to 2.
 *
 * Calling the cuda_update_state_after_solving function
 * ====================================================
 *
 * When calling the function, the blocks and threads must be set up correctly,
 * as shown in the following example code.
 *
 *   dim3  threads(1, 1, 1);
 *   dim3  blocks(1, 1, 1);
 *
 *   int const example_value = 256;
 *
 *   threads.x = min(n_fits, example_value);
 *   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
 *
 *   cuda_update_state_after_solving<<< blocks, threads >>>(
 *       n_fits,
 *       solution_info,
 *       finished,
 *       gpufit_states);
 *
 */
    
__global__ void cuda_update_state_after_solving(
    int const n_fits,
    int const * cublas_info,
    int const * finished,
    int * states)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
        return;

    if (finished[fit_index])
        return;

    if (cublas_info[fit_index] != 0)
        states[fit_index] = SINGULAR_HESSIAN;
}
    
/* Description of the cuda_check_for_convergence function
* =======================================================
*
* This function checks after each iteration whether the fits are converged or not.
* It also checks whether the set maximum number of iterations is reached.
*
* Parameters:
*
* finished: An input and output vector which allows the calculation to be skipped
*           for single fits.
*
* tolerance: The tolerance value for the convergence set by user.
*
* states: An output vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred. If the maximum
*         number of iterations is reached without converging, it is set to 1. If
*         the fit converged it keeps its initial value of 0.
*
* chi_squares: An input vector of chi-square values for multiple fits. Used for the
*              convergence check.
*
* prev_chi_squares: An input vector of chi-square values for multiple fits calculated
*                   in the previous iteration. Used for the convergence check.
*
* iteration: The value of the current iteration. It is compared to the value
*            of the maximum number of iteration set by user.
*
* max_n_iterations: The maximum number of iterations set by user.
*
* n_fits: The number of fits.
*
* Calling the cuda_check_for_convergence function
* ===============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_check_for_convergence<<< blocks, threads >>>(
*       finished,
*       tolerance,
*       states,
*       chi_squares,
*       prev_chi_squares,
*       iteration,
*       max_n_iterations,
*       n_fits);
*
*/

__global__ void cuda_check_for_convergence(
    int * finished,
    REAL const tolerance,
    int * states,
    REAL const * chi_squares,
    REAL const * prev_chi_squares,
    int const iteration,
    int const max_n_iterations,
    int const n_fits)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (finished[fit_index])
    {
        return;
    }

    int const fit_found
        = abs(chi_squares[fit_index] - prev_chi_squares[fit_index])
        < tolerance * max(1., chi_squares[fit_index]);

    int const max_n_iterations_reached = iteration == max_n_iterations - 1;

    if (fit_found)
    {
        finished[fit_index] = 1;
    }
    else if (max_n_iterations_reached)
    {
        states[fit_index] = MAX_ITERATION;
    }
}

/* Description of the cuda_evaluate_iteration function
* ====================================================
*
* This function evaluates the current iteration.
*   - It marks a fit as finished if a problem occured.
*   - It saves the needed number of iterations if a fit finished.
*   - It checks if all fits finished
*
* Parameters:
*
* all_finished: An output flag, that indicates whether all fits finished.
*
* n_iterations: An output vector of needed iterations for each fit.
*
* finished: An input and output vector which allows the evaluation to be skipped
*           for single fits
*
* iteration: The values of the current iteration.
*
* states: An input vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred.
*
* n_fits: The number of fits.
*
* Calling the cuda_evaluate_iteration function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_evaluate_iteration<<< blocks, threads >>>(
*       all_finished,
*       n_iterations,
*       finished,
*       iteration,
*       states,
*       n_fits);
*
*/

__global__ void cuda_evaluate_iteration(
    int * all_finished,
    int * n_iterations,
    int * finished,
    int const iteration,
    int const * states,
    int const n_fits)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (states[fit_index] != CONVERGED)
    {
        finished[fit_index] = 1;
    }

    if (finished[fit_index] && n_iterations[fit_index] == 0)
    {
        n_iterations[fit_index] = iteration + 1;
    }

    if (!finished[fit_index])
    {
        *all_finished = 0;
    }
}

/* Description of the cuda_prepare_next_iteration function
* ========================================================
*
* This function prepares the next iteration. It either updates previous
* chi-square values or sets currently calculated chi-square values and
* parameters to values calculated by the previous iteration. This function also
* updates lambda values.
*
* Parameters:
*
* lambdas: An output vector of values which control the step width by modifying
*          the diagonal elements of the hessian matrices.
*
* chi_squares: An input and output vector of chi-square values for multiple fits.
*
* prev_chi_squares: An input and output vector of chi-square values for multiple
*                   fits calculated in the previous iteration.
*
* parameters: An output vector of concatenated sets of model parameters.
*
* prev_parameters: An input vector of concatenated sets of model parameters
*                  calculated in the previous iteration.
*
* n_fits: The number of fits.
*
* n_parameters: The number of fitting curve parameters.
*
* Calling the cuda_prepare_next_iteration function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_prepare_next_iteration<<< blocks, threads >>>(
*       lambdas,
*       chi_squares,
*       prev_chi_squares,
*       parameters,
*       prev_parameters,
*       n_fits,
*       n_parameters);
*
*/

__global__ void cuda_prepare_next_iteration(
    REAL * lambdas,
    REAL * chi_squares,
    REAL * prev_chi_squares,
    REAL * parameters,
    REAL const * prev_parameters,
    int const n_fits,
    int const n_parameters)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (chi_squares[fit_index] < prev_chi_squares[fit_index])
    {
        lambdas[fit_index] *= 0.1f;
        prev_chi_squares[fit_index] = chi_squares[fit_index];
    }
    else
    {
        lambdas[fit_index] *= 10.;
        chi_squares[fit_index] = prev_chi_squares[fit_index];
        for (int iparameter = 0; iparameter < n_parameters; iparameter++)
        {
            parameters[fit_index * n_parameters + iparameter] = prev_parameters[fit_index * n_parameters + iparameter];
        }
    }
}
